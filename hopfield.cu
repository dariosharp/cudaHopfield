#include "hip/hip_runtime.h"
//nvcc -arch=sm_20 hopfield.cu -o hopfield
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/types.h>
#include <hip/hip_runtime_api.h>

#define sizeGrid 65535
#define sizeBlok 1024
#define sizeWarp 32


__global__ void training(int dimP, int nP, int *ps, float *ws){
	extern __shared__ float s[];
	int x;
	x = blockIdx.x*blockDim.x + threadIdx.x;
	for (int i = 0; i < nP; i++)	
		s[x] += (float)(ps[i*dimP+(x/dimP)]*ps[i*dimP+(x%dimP)]);
	s[((x/dimP)*dimP)+(x/dimP)] = 0.0f;
	__syncthreads();
	ws[x] = s[x]/nP;
}


int main(int argc, char *argv[]){
	int *patterns, *ps;
	float *ws, *weights;
	int nPatterns, dimPatterns;
	//printf("Insert number of patterns: ");
	//scanf("%d", &num);
	nPatterns = 2;
	dimPatterns = 4;

	int sizeP = dimPatterns*sizeof(int);
	int sizeW = dimPatterns*dimPatterns*sizeof(float);
	if ((patterns = (int*) malloc (sizeP*nPatterns)) == NULL ) return 1;
	if ((weights = (float*) malloc (sizeW)) == NULL ) return 1;
	hipMalloc ( &ps, (sizeP*nPatterns));
	hipMalloc ( &ws, (sizeW));
      
	for (int i = 0; i < nPatterns*dimPatterns; i++) {
		patterns[i] = rand() % 2 == 0 ? -1 : 1; 
	}
	hipMemcpy (ps, patterns, sizeP*nPatterns, hipMemcpyHostToDevice);

	for (int j = 0; j < nPatterns; j++){
		printf("[ ");
		for (int i = 0; i < dimPatterns; i++) {
			printf("%d ", patterns[j*dimPatterns + i]);
		}
		 printf("]\n");
	}
 
	dim3 GRID_DIM (1);//(int)((dimPatterns*dimPatterns)/sizeGrid)+1);
	dim3 BLOCK_DIM (dimPatterns*dimPatterns);
	training<<< GRID_DIM, BLOCK_DIM, dimPatterns*dimPatterns*sizeof(float) >>> (dimPatterns, nPatterns, ps, ws);
	hipDeviceSynchronize();
  
	hipMemcpy (weights, ws, sizeW, hipMemcpyDeviceToHost);	
   	printf("C:\n");
   	for(int i = 0; i < dimPatterns; i++){
      		printf("[ ");
      		for (int j = 0; j < dimPatterns; j++) {
         		printf("%.3f ", weights[i*dimPatterns+j]);
      		}
      		printf("]\n"); 
   	}	
}
