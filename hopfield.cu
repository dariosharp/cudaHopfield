//nvcc -arch=sm_20 hopfield.cu -o hopfield

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/types.h>
#include <hip/hip_runtime_api.h>

#define sizeGrid 65535
#define sizeBlok 1024
#define sizeWarp 32


__global__ void training(int dimP, int nP, int *ps, float *ws){
	extern __shared__ float s[];
	int x;
	x = blockIdx.x*blockDim.x + threadIdx.x;
	for (int i = 0; i < nP; i++)	
		s[x] += (float)((2*ps[i*dimP+(x/dimP)]-1)*(2*ps[i*dimP+(x%dimP)]-1));
	s[((x/dimP)*dimP)+(x/dimP)] = 0.0f;
	ws[x] = s[x]/nP;
}

__global__ void hopActivation(int dimP, float *ws, int *pt, int *at)
{
        extern __shared__ float sdata [];
        int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int wpS = sizeWarp;
	if (dimP < 32)
		wpS = dimP; 
        int wid = tid / wpS;
        int lane= tid % wpS;
        if (wid < wpS ){
                int start_neuron = (wid*dimP);
                int end_neuron = ((wid+1)*dimP);
                sdata[threadIdx.x]=0;
                for(int i=start_neuron+lane;i<end_neuron;i+=32)
                        sdata[threadIdx.x]+= ws[i] * (2*pt[i % dimP ] -1);
		__syncthreads();
                if (lane + 16 < dimP) sdata[threadIdx.x] += sdata[threadIdx.x+16]; __syncthreads();
		if (lane +  8 < dimP) sdata[threadIdx.x] += sdata[threadIdx.x+ 8]; __syncthreads();
                if (lane +  4 < dimP) sdata[threadIdx.x] += sdata[threadIdx.x+ 4]; __syncthreads();
                if (lane +  2 < dimP) sdata[threadIdx.x] += sdata[threadIdx.x+ 2]; __syncthreads();
                if (lane +  1 < dimP) sdata[threadIdx.x] += sdata[threadIdx.x+ 1];
                if (lane == 0)
	        	at[wid] = ((sdata[threadIdx.x] > 0) - (sdata[threadIdx.x] < 0)+1)/2;
        }
}



float * lState (int nPatterns, int dimPattern, int *patterns){
	int *ps;
	float *weights, *ws;
	int sizeP = dimPattern*sizeof(int);
	int sizeW = dimPattern*dimPattern*sizeof(float);

	if ((weights = (float*) malloc (sizeW)) == NULL ) return NULL;
	if ( hipSuccess != hipMalloc ( &ps, (sizeP*nPatterns))) return NULL;
	if ( hipSuccess != hipMalloc ( &ws, (sizeW))) return NULL;
	if ( hipSuccess != hipMemcpy (ps, patterns, sizeP*nPatterns, hipMemcpyHostToDevice)) return NULL;

	dim3 GRID_DIM (1);
	dim3 BLOCK_DIM (dimPattern*dimPattern);
	training<<< GRID_DIM, BLOCK_DIM, dimPattern*dimPattern*sizeof(float) >>> (dimPattern, nPatterns, ps, ws);
  
	if (hipSuccess != hipMemcpy (weights, ws, sizeW, hipMemcpyDeviceToHost)) return NULL;
   	return weights;
}


int * actFunc(int dP, int *pattern, float *weight){
	float *ws;
	int *pt, *activation, *at;
	if ( (activation = (int *) malloc (dP*sizeof(int))) == NULL) return NULL;
	if (hipSuccess != hipMalloc (&ws, dP*dP*sizeof(float))) return NULL;
	if (hipSuccess != hipMalloc (&pt, dP*sizeof(int))) return NULL;
	if (hipSuccess != hipMalloc (&at, dP*sizeof(int))) return NULL;
	if ( hipSuccess != hipMemcpy (ws, weight, dP*dP*sizeof(float), hipMemcpyHostToDevice)) return NULL;
	if ( hipSuccess != hipMemcpy (pt, pattern, dP*sizeof(int), hipMemcpyHostToDevice)) return NULL;

	dim3 GRID_DIM (1);
	dim3 BLOCK_DIM (dP*dP);
	hopActivation<<< GRID_DIM, BLOCK_DIM, dP*dP*sizeof(float) >>> (dP, ws, pt, at);

  	if (hipSuccess != hipMemcpy (activation, at, dP*sizeof(int), hipMemcpyDeviceToHost)) return NULL;
	return activation;
}


int main(int argc, char *argv[]){
	int nPatterns, dimPattern;
	int * patterns;

	nPatterns = 2;
	dimPattern = 7;
	if ((patterns = (int*) malloc (dimPattern*nPatterns*sizeof(int))) == NULL ) return 1;

	for (int i = 0; i < nPatterns*dimPattern; i++) {
		patterns[i] = rand() % 2; 
	}
	for (int j = 0; j < nPatterns; j++){
		printf("[ ");
		for (int i = 0; i < dimPattern; i++) {
			printf("%d ", patterns[j*dimPattern + i]);
		}
		 printf("]\n");
	}

	float * weights = lState(nPatterns, dimPattern, patterns);
	if (weights == NULL){
		printf("Error on Learning\n");
		return 1;
	}

	printf("Weights:\n");
   	for(int i = 0; i < dimPattern; i++){
      		printf("[ ");
      		for (int j = 0; j < dimPattern; j++) {
         		printf("%.3f ", weights[i*dimPattern+j]);
      		}
      		printf("]\n"); 
   	}
	
	int * epat = (int *)malloc (dimPattern*sizeof(int));
	epat[0] = 1;
	epat[1] = 0;
	epat[2] = 1;
	epat[3] = 0;
	epat[4] = 1;
	epat[5] = 1;
	epat[6] = 0;

	int * activation = actFunc(dimPattern, epat, weights);
	if (activation == NULL){
		printf("Error on Activarion\n");
		return 1;
	}
	printf("activation [");
	for (int i = 0; i < dimPattern; i++)
		printf("%i ", activation[i]);
	printf("]\n");
}
